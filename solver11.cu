#include "hip/hip_runtime.h"
#include "solver11.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstring>
#include "cudahelper11.h"
#include <iostream>

static dim3 createDim2DFrom1D( unsigned int size, unsigned int maxSize )
{
	dim3 dim;
	if( size > maxSize )
	{
		dim.x = maxSize;
		dim.y = size / maxSize;
	}
	else
	{
		dim.x = size;
		dim.y = 1;
	}
	dim.z = 1;

	return dim;
}

Solver11::Solver11()
{
}

Solver11::~Solver11()
{
}

void Solver11::solve()
{
	hipDeviceSetCacheConfig( hipFuncCachePreferShared );

	unsigned int ip = np / ns;
	real h = 2 * L / ( np - 1 );
	real dt = h / 4.0;
	real l = dt / h;
	real l2 = l * l;
	unsigned int nsteps = ip / 2;
	unsigned int ndom = 2 * ns - 1;
	unsigned int gpudom = 2 * ns;

	dim3 gridSize = createDim2DFrom1D( gpudom, 1 << 16 );
	unsigned int threads = ip / 2;
	dim3 blockSize = createDim2DFrom1D( threads, 1024 );
	unsigned int syncThreads = threads / 4;
	dim3 syncBlockSize = createDim2DFrom1D( syncThreads, 1024 );
	unsigned int shmem = ( ip + 4 ) * sizeof( real );

	unsigned int kmax = ceil( T / ( nsteps * dt ) );
	solution = std::vector<double>( np );
	error = std::vector<double>( kmax );

	std::vector<real> z( gpudom * ip );
	std::vector<real> w( gpudom * ip );

	unsigned int bufSize = gpudom * ip * sizeof( real );

	real2* d_z;
	real2* d_w;
	hipMalloc( &d_z, bufSize );
	hipMalloc( &d_w, bufSize );

	std::vector<real> x( np );
	std::vector<real> f( np );
	std::vector<real> g( np );
	for( unsigned int i = 0; i < np; ++i )
	{
		x[ i ] = -L + i * h;
		f[ i ] = u0( x[ i ] );
		g[ i ] = u1( x[ i ] );
	}
	for( unsigned int n = 0; n < ndom; ++n )
	{
		memcpy( &w[ n * ip ], &f[ n * ip / 2 ], ip * sizeof( real ) );
		memcpy( &z[ n * ip ], &g[ n * ip / 2 ], ip * sizeof( real ) );
	}

	hipMemcpy( d_w, w.data(), bufSize, hipMemcpyHostToDevice );
	hipMemcpy( d_z, z.data(), bufSize, hipMemcpyHostToDevice );

	f.clear();
	g.clear();

	CudaHelper11::calculateFirstStep<<<gridSize, blockSize, shmem>>>(
			dt,	h, ip / 2, d_w, d_z, d_z );

	CudaHelper11::synchronizeResults<<<gridSize, syncBlockSize>>>(
			ip / 2,
			d_z,
			d_w );

	for( unsigned int k = 0; k < kmax; ++k )
	{
		CudaHelper11::calculateNSteps<<<gridSize, blockSize, shmem>>>(
				nsteps,	l2, 2.0 * ( 1.0 - l2 ), ip / 2, d_z, d_w );

		CudaHelper11::synchronizeResults<<<gridSize, syncBlockSize>>>(
				ip / 2,
				d_z,
				d_w );

		if( k > 0 )
		{
			calculateError( z.data(), k - 1, ip, nsteps, dt, h, x.data() );
		}

		hipMemcpy( z.data(), d_z, bufSize, hipMemcpyDeviceToHost );
	}

	calculateError( z.data(), kmax - 1, ip, nsteps, dt, h, x.data() );

	hipFree( d_z );
	hipFree( d_w );
}

const char* Solver11::getName() const
{
	return "Solver11";
}

void Solver11::calculateError( real* z, unsigned int k,
		unsigned int ip, unsigned int nsteps, real dt, real h, real* x )
{
	for( unsigned int n = 0; n < ns; ++n )
	{
		memcpy( &solution[ n * ip ], &z[ 2 * n * ip ], ip * sizeof( real ) );
	}

	double t = ( ( k + 1 ) * nsteps + 1 ) * dt;
	double l2err = 0.0;
	for( unsigned int i = 0; i < np; ++i )
	{
		double err = sol( x[ i ], t ) - solution[ i ];
		l2err += err * err;
	}

	error[ k ] = sqrt( h * l2err );
}
