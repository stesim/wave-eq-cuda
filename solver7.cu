#include "hip/hip_runtime.h"
#include "solver7.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstring>
#include "cudahelper7.h"
#include <iostream>

Solver7::Solver7()
{
}

Solver7::~Solver7()
{
}

void Solver7::solve()
{
	//hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

	unsigned int ip = np / ns;
	double h = 2 * L / ( np - 1 );
	double dt = h / 4.0;
	double l = dt / h;
	double l2 = l * l;
	unsigned int nsteps = ip / 4;
	unsigned int ndom = 2 * ns - 1;
	unsigned int gpudom = 2 * ns;
	unsigned int blocks = gpudom / threads;

	unsigned int kmax = ceil( T / ( nsteps * dt ) );
	solution = std::vector<double>( np );
	error = std::vector<double>( kmax );

	std::vector<double> z( gpudom * ip );
	std::vector<double> w( gpudom * ip );
	std::vector<double> u( gpudom * ip );

	unsigned int bufSize = gpudom * ip * sizeof( double );

	double2* d_z;
	double2* d_w;
	double2* d_u;
	hipMalloc( &d_z, bufSize );
	hipMalloc( &d_w, bufSize );
	hipMalloc( &d_u, bufSize );

	std::vector<double> x( np );
	std::vector<double> f( np );
	std::vector<double> g( np );
	for( unsigned int i = 0; i < np; ++i )
	{
		x[ i ] = -L + i * h;
		f[ i ] = u0( x[ i ] );
		g[ i ] = u1( x[ i ] );
	}
	for( unsigned int n = 0; n < blocks; ++n )
	{
		for( unsigned int j = 0; j < threads; ++j )
		{
			if( n == blocks - 1 && j == threads - 1 )
			{
				continue;
			}

			unsigned int base = n * threads * ip + 2 * j;
			unsigned int srcBase = ( n * threads + j ) * ip / 2;
			for( unsigned int i = 0; i < ip / 2; ++i )
			{
				w[ base + 2 * i * threads ] = f[ srcBase + 2 * i ];
				w[ base + 2 * i * threads + 1 ] = f[ srcBase + 2 * i + 1 ];
				u[ base + 2 * i * threads ] = g[ srcBase + 2 * i ];
				u[ base + 2 * i * threads + 1 ] = g[ srcBase + 2 * i + 1 ];
			}
		}
	}

	hipMemcpy( d_w, w.data(), bufSize, hipMemcpyHostToDevice );
	hipMemcpy( d_u, u.data(), bufSize, hipMemcpyHostToDevice );

	f.clear();
	g.clear();

	SpDiaMat matInner = gpuAllocFDMatrixInner( l2, ip );
	SpDiaMat matLeft = gpuAllocFDMatrixLeft( l2, ip );
	SpDiaMat matRight = gpuAllocFDMatrixRight( l2, ip );

	CudaHelper7::calculateFirstStep<<<blocks, threads>>>(
			dt,	h, ip, matInner, matLeft, matRight,	d_w, d_u, d_z );

	CudaHelper7::synchronizeResults<<<blocks, threads>>>(
			ip,
			d_z,
			d_w );

		/*
		hipMemcpy(	z.data(), d_z, bufSize,	hipMemcpyDeviceToHost );
		for( unsigned int n = 0; n < blocks; ++n )
		{
			for( unsigned int j = 0; j < threads; j += 2 )
			{
				if( n == blocks - 1 && j == threads - 1 )
				{
					continue;
				}

				unsigned int base = n * threads * ip + 2 * j;
				unsigned int dstBase = ( n * threads + j ) * ip / 2;
				for( unsigned int i = 0; i < ip / 2; ++i )
				{
					solution[ dstBase + 2 * i ] = z[ base + 2 * i * threads ];
					solution[ dstBase + 2 * i + 1 ] = z[ base + 2 * i * threads + 1 ];
				}
			}
		}
		for( unsigned int i = 0; i < 32; ++i )
		{
			std::cout << solution[ i ] << std::endl;
		}
		*/

	for( unsigned int k = 0; k < kmax; ++k )
	{
		CudaHelper7::calculateNSteps<<<blocks, threads/*, threads * ip * sizeof( double )*/>>>(
				nsteps,	2.0 * ( 1.0 - l2 ),	ip,	matInner, matLeft, matRight,
				d_z, d_w, d_u );

		double2* swap;
		switch( nsteps % 3 )
		{
			case 1:
				swap = d_w;
				d_w = d_z;
				d_z = d_u;
				d_u = swap;
				break;
			case 2:
				swap = d_z;
				d_z = d_w;
				d_w = d_u;
				d_u = swap;
		}

		CudaHelper7::synchronizeResults<<<blocks, threads>>>(
				ip,
				d_z,
				d_w );

		hipDeviceSynchronize();

		hipMemcpy(
				z.data(),
				d_z,
				bufSize,
				hipMemcpyDeviceToHost );

		for( unsigned int n = 0; n < blocks; ++n )
		{
			for( unsigned int j = 0; j < threads; j += 2 )
			{
				if( n == blocks - 1 && j == threads - 1 )
				{
					continue;
				}

				unsigned int base = n * threads * ip + 2 * j;
				unsigned int dstBase = ( n * threads + j ) * ip / 2;
				for( unsigned int i = 0; i < ip / 2; ++i )
				{
					solution[ dstBase + 2 * i ] = z[ base + 2 * i * threads ];
					solution[ dstBase + 2 * i + 1 ] = z[ base + 2 * i * threads + 1 ];
				}
			}
		}

		double t = ( ( k + 1 ) * nsteps + 1 ) * dt;
		double l2err = 0.0;
		for( unsigned int i = 0; i < np; ++i )
		{
			double err = sol( x[ i ], t ) - solution[ i ];
			l2err += err * err;
		}
		error[ k ] = sqrt( h * l2err );
	}

	solution = z;

	hipFree( d_z );
	hipFree( d_w );
	hipFree( d_u );
	gpuFreeFDMatrix( matInner );
	gpuFreeFDMatrix( matLeft );
	gpuFreeFDMatrix( matRight );
}

const char* Solver7::getName() const
{
	return "Solver7";
}

SpDiaMat Solver7::gpuAllocFDMatrixInner( double l2, unsigned int ip )
{
	unsigned int diagSize = ip - 1;

	SpDiaMat mat;
	mat.n = ip;
	mat.diags = 2;
	hipMalloc( &mat.offsets, 2 * sizeof( int ) );
	hipMalloc( &mat.values, 2 * diagSize * sizeof( double ) );

	static const int offsets[] = { 1, -1 };
	double* values = new double[ 2 * diagSize * sizeof( double ) ];

	for( unsigned int i = 0; i < diagSize; ++i )
	{
		values[ i ] = l2;
		values[ diagSize + i ] = l2;
	}

	hipMemcpy( const_cast<int*>( mat.offsets ), offsets, 2 * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( const_cast<double*>( mat.values ), values, 2 * diagSize * sizeof( double ), hipMemcpyHostToDevice );

	delete[] values;
	return mat;
}

SpDiaMat Solver7::gpuAllocFDMatrixLeft( double l2, unsigned int ip )
{
	unsigned int diagSize = ip - 1;

	SpDiaMat mat;
	mat.n = ip;
	mat.diags = 2;
	hipMalloc( &mat.offsets, 2 * sizeof( int ) );
	hipMalloc( &mat.values, 2 * diagSize * sizeof( double ) );

	static const int offsets[] = { 1, -1 };
	double* values = new double[ 2 * diagSize * sizeof( double ) ];

	for( unsigned int i = 0; i < diagSize; ++i )
	{
		values[ i ] = ( i != 0 ? l2 : 2.0 * l2 );
		values[ diagSize + i ] = l2;
	}

	hipMemcpy( const_cast<int*>( mat.offsets ), offsets, 2 * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( const_cast<double*>( mat.values ), values, 2 * diagSize * sizeof( double ), hipMemcpyHostToDevice );

	delete[] values;
	return mat;
}

SpDiaMat Solver7::gpuAllocFDMatrixRight( double l2, unsigned int ip )
{
	unsigned int diagSize = ip - 1;

	SpDiaMat mat;
	mat.n = ip;
	mat.diags = 2;
	hipMalloc( &mat.offsets, 2 * sizeof( int ) );
	hipMalloc( &mat.values, 2 * diagSize * sizeof( double ) );

	static const int offsets[] = { 1, -1 };
	double* values = new double[ 2 * diagSize * sizeof( double ) ];

	for( unsigned int i = 0; i < diagSize; ++i )
	{
		values[ i ] = l2;
		values[ diagSize + i ] = ( i != diagSize - 1 ? l2 : 2.0 * l2 );
	}

	hipMemcpy( const_cast<int*>( mat.offsets ), offsets, 2 * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( const_cast<double*>( mat.values ), values, 2 * diagSize * sizeof( double ), hipMemcpyHostToDevice );

	delete[] values;
	return mat;
}

void Solver7::gpuFreeFDMatrix( SpDiaMat& mat )
{
	hipFree( const_cast<int*>( mat.offsets ) );
	mat.offsets = NULL;
	hipFree( const_cast<double*>( mat.values ) );
	mat.values = NULL;
}
