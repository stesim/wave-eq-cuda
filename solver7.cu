#include "hip/hip_runtime.h"
#include "solver7.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstring>
#include "cudahelper7.h"
#include <iostream>

Solver7::Solver7()
{
}

Solver7::~Solver7()
{
}

void Solver7::solve()
{
	hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

	unsigned int ip = np / ns;
	double h = 2 * L / ( np - 1 );
	double dt = h / 4.0;
	double l = dt / h;
	double l2 = l * l;
	unsigned int nsteps = ip / 2;
	unsigned int ndom = 2 * ns - 1;
	unsigned int gpudom = 2 * ns;
	unsigned int blocks = gpudom / threads / 2;

	std::vector<double> z( gpudom * ip );
	std::vector<double> w( gpudom * ip );
	std::vector<double> u( gpudom * ip );

	unsigned int bufSize = gpudom * ip * sizeof( double );

	double* d_z;
	double* d_w;
	double* d_u;
	hipMalloc( &d_z, bufSize );
	hipMalloc( &d_w, bufSize );
	hipMalloc( &d_u, bufSize );

	std::vector<double> x( np );
	std::vector<double> f( np );
	std::vector<double> g( np );
	for( unsigned int i = 0; i < np; ++i )
	{
		x[ i ] = -L + i * h;
		f[ i ] = u0( x[ i ] );
		g[ i ] = u1( x[ i ] );
	}
	/*
	for( unsigned int i = 0; i < ndom; ++i )
	{
		unsigned int pos = i * ip;
		memcpy( &w[ pos ], &f[ pos / 2 ], ip * sizeof( double ) );
		memcpy( &u[ pos ], &g[ pos / 2 ], ip * sizeof( double ) ); 
	}
	*/
	for( unsigned int n = 0; n < blocks; ++n )
	{
		for( unsigned int k = 0; k < threads; ++k )
		{
			unsigned int base = 2 * ( n * threads * ip + k );
			unsigned int srcBase = ( n * threads + k ) * ip;
			for( unsigned int i = 0; i < ip; ++i )
			{
				w[ base + i * 2 * threads ] = f[ srcBase + i ];
				u[ base + i * 2 * threads ] = g[ srcBase + i ];

				if( n != blocks - 1 || k != threads - 1 )
				{
					w[ base + i * 2 * threads + 1 ] = f[ srcBase + ip / 2 + i ];
					u[ base + i * 2 * threads + 1 ] = g[ srcBase + ip / 2 + i ];
				}
			}
		}
	}
	for( unsigned int i = 0; i < 32; ++i )
	{
		std::cout << w[ i ] << std::endl;
	}
		/*
		for( unsigned int n = 0; n < blocks; ++n )
		{
			for( unsigned int k = 0; k < threads; ++k )
			{
				unsigned int base = 2 * ( n * threads * ip + k );
				unsigned int dstBase = ( n * threads + k ) * ip;
				for( unsigned int i = 0; i < ip / 2; ++i )
				{
					g[ dstBase + i ] = w[ base + i * 2 * threads ];

					if( n != blocks - 1 || k != threads - 1 )
					{
						g[ dstBase + ip / 2 + i ] = w[ base + i * 2 * threads + 1 ];
					}
				}
			}
		}
		unsigned int diffs = 0;
		for( unsigned int i = 0; i < np; ++i )
		{
			if( f[ i ]  != g[ i ] )
			{
				std::cout << i << std::endl;
				//break;
				++diffs;
			}
		}
		std::cout << diffs << std::endl;
		*/

	hipMemcpy( d_w, w.data(), bufSize, hipMemcpyHostToDevice );
	hipMemcpy( d_u, u.data(), bufSize, hipMemcpyHostToDevice );

	f.clear();
	g.clear();

	SpDiaMat matInner = gpuAllocFDMatrixInner( l2, ip );
	SpDiaMat matLeft = gpuAllocFDMatrixLeft( l2, ip );
	SpDiaMat matRight = gpuAllocFDMatrixRight( l2, ip );

	CudaHelper7::calculateFirstStep<<<blocks, threads>>>(
			dt,	h, ip, matInner, matLeft, matRight,	(double2*)d_w, (double2*)d_u, (double2*)d_z );

	CudaHelper7::synchronizeResults<<<blocks, threads>>>(
			ip,
			(double2*)d_z,
			(double2*)d_w );

	/*
	hipMemcpy( z.data(), d_z, bufSize, hipMemcpyDeviceToHost );
	for( unsigned int i = 0; i < 32; ++i )
	{
		std::cout << z[ i ] << std::endl;
	}
	*/

	solution = std::vector<double>( np );

	unsigned int kmax = ceil( T / ( nsteps * dt ) );
	error = std::vector<double>( kmax );

	for( unsigned int k = 0; k < kmax; ++k )
	{
		CudaHelper7::calculateNSteps<<<blocks, threads>>>(
				nsteps,	2.0 * ( 1.0 - l2 ),	ip,	matInner, matLeft, matRight,
				(double2*)d_z, (double2*)d_w, (double2*)d_u );

		double* swap;
		switch( nsteps % 3 )
		{
			case 1:
				swap = d_w;
				d_w = d_z;
				d_z = d_u;
				d_u = swap;
				break;
			case 2:
				swap = d_z;
				d_z = d_w;
				d_w = d_u;
				d_u = swap;
		}

		CudaHelper7::synchronizeResults<<<blocks, threads>>>(
				ip,
				(double2*)d_z,
				(double2*)d_w );

		/*
		CudaHelper7::copyResultsLinear<<<blocks, threads>>>(
				ip, d_z, d_solution );

		hipMemcpy( solution.data(), d_solution, np * sizeof( double ), hipMemcpyDeviceToHost );
		*/

		hipMemcpy(
				z.data(),
				d_z,
				bufSize,
				hipMemcpyDeviceToHost );

		/*
		for( unsigned int i = 0; i < ns; ++i )
		{
			memcpy(
					&solution[ i * ip ],
					&z[ 2 * i * ip ],
					ip * sizeof( double ) );
		}
		*/
		for( unsigned int n = 0; n < blocks; ++n )
		{
			for( unsigned int k = 0; k < threads; ++k )
			{
				unsigned int base = 2 * ( n * threads * ip + k );
				unsigned int dstBase = ( n * threads + k ) * ip;
				for( unsigned int i = 0; i < ip / 2; ++i )
				{
					solution[ dstBase + i ] = z[ base + i * 2 * threads ];

					if( n != blocks - 1 || k != threads - 1 )
					{
						solution[ dstBase + ip / 2 + i ] = z[ base + i * 2 * threads + 1 ];
					}
				}
			}
		}

		double t = ( ( k + 1 ) * nsteps + 1 ) * dt;
		double l2err = 0.0;
		for( unsigned int i = 0; i < np; ++i )
		{
			double err = sol( x[ i ], t ) - solution[ i ];
			l2err += err * err;
		}
		error[ k ] = sqrt( h * l2err );
	}

	solution = z;

	hipFree( d_z );
	hipFree( d_w );
	hipFree( d_u );
	gpuFreeFDMatrix( matInner );
	gpuFreeFDMatrix( matLeft );
	gpuFreeFDMatrix( matRight );
}

SpDiaMat Solver7::gpuAllocFDMatrixInner( double l2, unsigned int ip )
{
	unsigned int diagSize = ip - 1;

	SpDiaMat mat;
	mat.n = ip;
	mat.diags = 2;
	hipMalloc( &mat.offsets, 2 * sizeof( int ) );
	hipMalloc( &mat.values, 2 * diagSize * sizeof( double ) );

	static const int offsets[] = { 1, -1 };
	double* values = new double[ 2 * diagSize * sizeof( double ) ];

	for( unsigned int i = 0; i < diagSize; ++i )
	{
		values[ i ] = l2;
		values[ diagSize + i ] = l2;
	}

	hipMemcpy( const_cast<int*>( mat.offsets ), offsets, 2 * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( const_cast<double*>( mat.values ), values, 2 * diagSize * sizeof( double ), hipMemcpyHostToDevice );

	delete[] values;
	return mat;
}

SpDiaMat Solver7::gpuAllocFDMatrixLeft( double l2, unsigned int ip )
{
	unsigned int diagSize = ip - 1;

	SpDiaMat mat;
	mat.n = ip;
	mat.diags = 2;
	hipMalloc( &mat.offsets, 2 * sizeof( int ) );
	hipMalloc( &mat.values, 2 * diagSize * sizeof( double ) );

	static const int offsets[] = { 1, -1 };
	double* values = new double[ 2 * diagSize * sizeof( double ) ];

	for( unsigned int i = 0; i < diagSize; ++i )
	{
		values[ i ] = ( i != 0 ? l2 : 2.0 * l2 );
		values[ diagSize + i ] = l2;
	}

	hipMemcpy( const_cast<int*>( mat.offsets ), offsets, 2 * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( const_cast<double*>( mat.values ), values, 2 * diagSize * sizeof( double ), hipMemcpyHostToDevice );

	delete[] values;
	return mat;
}

SpDiaMat Solver7::gpuAllocFDMatrixRight( double l2, unsigned int ip )
{
	unsigned int diagSize = ip - 1;

	SpDiaMat mat;
	mat.n = ip;
	mat.diags = 2;
	hipMalloc( &mat.offsets, 2 * sizeof( int ) );
	hipMalloc( &mat.values, 2 * diagSize * sizeof( double ) );

	static const int offsets[] = { 1, -1 };
	double* values = new double[ 2 * diagSize * sizeof( double ) ];

	for( unsigned int i = 0; i < diagSize; ++i )
	{
		values[ i ] = l2;
		values[ diagSize + i ] = ( i != diagSize - 1 ? l2 : 2.0 * l2 );
	}

	hipMemcpy( const_cast<int*>( mat.offsets ), offsets, 2 * sizeof( int ), hipMemcpyHostToDevice );
	hipMemcpy( const_cast<double*>( mat.values ), values, 2 * diagSize * sizeof( double ), hipMemcpyHostToDevice );

	delete[] values;
	return mat;
}

void Solver7::gpuFreeFDMatrix( SpDiaMat& mat )
{
	hipFree( const_cast<int*>( mat.offsets ) );
	mat.offsets = NULL;
	hipFree( const_cast<double*>( mat.values ) );
	mat.values = NULL;
}
